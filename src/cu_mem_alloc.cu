#include "cu_mem_alloc.h"
#include "hip/hip_runtime.h"
#include <cu_runtime_error.h>

namespace gsy {
    template<>
    void* CuMemoryAdapter<DeviceMemory>::allocate(std::size_t size) {
        void* ptr = nullptr;
        hipError_t err = hipMalloc(&ptr, size);
        if(err != hipSuccess) {
            throw CuRuntimeError(err);
        }
        return ptr;
    }

    template<>
    void CuMemoryAdapter<DeviceMemory>::deallocate(void* ptr) noexcept {
        hipFree(ptr);
    }

    template<>
    void* CuMemoryAdapter<UnifiedMemory>::allocate(std::size_t size) {
        void* ptr = nullptr;
        hipError_t err = hipMallocManaged(&ptr, size);
        if(err != hipSuccess) {
            throw CuRuntimeError(err);
        }
        return ptr;
    }

    template<>
    void CuMemoryAdapter<UnifiedMemory>::deallocate(void* ptr) noexcept {
        hipFree(ptr);
    }
}